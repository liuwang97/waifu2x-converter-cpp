
#include <hip/hip_runtime.h>
/* -*- mode: c++ -*- */

#define UNROLL9(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\
	F(8);					\


#define UNROLL8x3x3(F)				\
	F(0,0,0);				\
	F(0,0,1);				\
	F(0,0,2);				\
	F(0,1,0);				\
	F(0,1,1);				\
	F(0,1,2);				\
	F(0,2,0);				\
	F(0,2,1);				\
	F(0,2,2);				\
						\
	F(1,0,0);				\
	F(1,0,1);				\
	F(1,0,2);				\
	F(1,1,0);				\
	F(1,1,1);				\
	F(1,1,2);				\
	F(1,2,0);				\
	F(1,2,1);				\
	F(1,2,2);				\
						\
	F(2,0,0);				\
	F(2,0,1);				\
	F(2,0,2);				\
	F(2,1,0);				\
	F(2,1,1);				\
	F(2,1,2);				\
	F(2,2,0);				\
	F(2,2,1);				\
	F(2,2,2);				\
						\
	F(3,0,0);				\
	F(3,0,1);				\
	F(3,0,2);				\
	F(3,1,0);				\
	F(3,1,1);				\
	F(3,1,2);				\
	F(3,2,0);				\
	F(3,2,1);				\
	F(3,2,2);				\
						\
	F(4,0,0);				\
	F(4,0,1);				\
	F(4,0,2);				\
	F(4,1,0);				\
	F(4,1,1);				\
	F(4,1,2);				\
	F(4,2,0);				\
	F(4,2,1);				\
	F(4,2,2);				\
						\
	F(5,0,0);				\
	F(5,0,1);				\
	F(5,0,2);				\
	F(5,1,0);				\
	F(5,1,1);				\
	F(5,1,2);				\
	F(5,2,0);				\
	F(5,2,1);				\
	F(5,2,2);				\
						\
	F(6,0,0);				\
	F(6,0,1);				\
	F(6,0,2);				\
	F(6,1,0);				\
	F(6,1,1);				\
	F(6,1,2);				\
	F(6,2,0);				\
	F(6,2,1);				\
	F(6,2,2);				\
						\
	F(7,0,0);				\
	F(7,0,1);				\
	F(7,0,2);				\
	F(7,1,0);				\
	F(7,1,1);				\
	F(7,1,2);				\
	F(7,2,0);				\
	F(7,2,1);				\
	F(7,2,2);				\

#define UNROLL8(F)				\
	F(0);					\
	F(1);					\
	F(2);					\
	F(3);					\
	F(4);					\
	F(5);					\
	F(6);					\
	F(7);					\


#define UNROLL8x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\


#define UNROLL10x3(F)				\
	F(0,0);					\
	F(0,1);					\
	F(0,2);					\
	F(0,3);					\
	F(0,4);					\
	F(0,5);					\
	F(0,6);					\
	F(0,7);					\
	F(0,8);					\
	F(0,9);					\
						\
	F(1,0);					\
	F(1,1);					\
	F(1,2);					\
	F(1,3);					\
	F(1,4);					\
	F(1,5);					\
	F(1,6);					\
	F(1,7);					\
	F(1,8);					\
	F(1,9);					\
						\
	F(2,0);					\
	F(2,1);					\
	F(2,2);					\
	F(2,3);					\
	F(2,4);					\
	F(2,5);					\
	F(2,6);					\
	F(2,7);					\
	F(2,8);					\
	F(2,9);					\


#define BLOCK_SIZE 8

extern "C" __global__ void
filter(const float * __restrict__ packed_input,
       int nInputPlanes,
       float * __restrict__ packed_output,
       int nOutputPlanes,
       const float * __restrict__ biases,
       unsigned int hsz,
       unsigned int wsz,
       const float * __restrict__ weight)
{
	extern __shared__ float shared_buf[];

	unsigned int yi = blockIdx.x;

	size_t in_step = wsz * nInputPlanes;
	const float *inp = packed_input;
	inp += yi * in_step;

	const float *in0p = inp - in_step;
	if (yi == 0) {
		in0p = inp;
	}
	const float *in1p = inp;

	const float *in2p = inp + in_step;
	if (yi == wsz-1) {
		in2p = inp;
	}

	const float *in01 = in0p;
	const float *in11 = in1p;
	const float *in21 = in2p;

	float *shared_ptr = shared_buf;
	float *in_block0_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block1_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);
	float *in_block2_base = shared_ptr;
	shared_ptr += nInputPlanes*(BLOCK_SIZE+2);

	float *in_block0 = in_block0_base + nInputPlanes;
	float *in_block1 = in_block1_base + nInputPlanes;
	float *in_block2 = in_block2_base + nInputPlanes;
	int lid = threadIdx.x;
	float bv = biases[lid];

	for (int xi0=0; xi0<wsz; xi0+=BLOCK_SIZE) {

		/*for (unsigned int op=0; op<nOutputPlanes; op++) thread */
		{
			int op = lid;
			int rem = wsz - xi0;
			__syncthreads();
			if (lid < nInputPlanes) {
				int bi;
				for (bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0 + bi;
					if (xi == wsz) {
						break;
					}

					/* load to shared */
					in_block0[bi*nInputPlanes + lid] = in01[xi*nInputPlanes + lid];
					in_block1[bi*nInputPlanes + lid] = in11[xi*nInputPlanes + lid];
					in_block2[bi*nInputPlanes + lid] = in21[xi*nInputPlanes + lid];
				}

				{
					int xi = xi0 + bi;
					if (xi == wsz) {
						in_block0[bi*(int)nInputPlanes + lid] = in01[(xi-1)*(int)nInputPlanes + lid];
						in_block1[bi*(int)nInputPlanes + lid] = in11[(xi-1)*(int)nInputPlanes + lid];
						in_block2[bi*(int)nInputPlanes + lid] = in21[(xi-1)*(int)nInputPlanes + lid];
					} else {
						in_block0[bi*(int)nInputPlanes + lid] = in01[xi*(int)nInputPlanes + lid];
						in_block1[bi*(int)nInputPlanes + lid] = in11[xi*(int)nInputPlanes + lid];
						in_block2[bi*(int)nInputPlanes + lid] = in21[xi*(int)nInputPlanes + lid];
					}
				}

				{
					int xi = xi0-1;
					if (xi == -1) {
						in_block0[-1*(int)nInputPlanes + (int)lid] = in01[lid];
						in_block1[-1*(int)nInputPlanes + (int)lid] = in11[lid];
						in_block2[-1*(int)nInputPlanes + (int)lid] = in21[lid];
					} else {
						in_block0[-1*(int)nInputPlanes + (int)lid] = in01[xi*(int)nInputPlanes + lid];
						in_block1[-1*(int)nInputPlanes + (int)lid] = in11[xi*(int)nInputPlanes + lid];
						in_block2[-1*(int)nInputPlanes + (int)lid] = in21[xi*(int)nInputPlanes + lid];
					}
				}
			}
			__syncthreads();

			if (rem >= BLOCK_SIZE) {
#define DECL_PTR(y,x)		float *p##y##x = &in_block##y[nInputPlanes * (x-1)];

				UNROLL10x3(DECL_PTR);

				float sum0 = 0;
				float sum1 = 0;
				float sum2 = 0;
				float sum3 = 0;

				float sum4 = 0;
				float sum5 = 0;
				float sum6 = 0;
				float sum7 = 0;

				{
					const float *w0 = weight + lid;

					for (int ip = 0; ip < nInputPlanes; ip++) {
#define LOAD_INPUT2(y,x)			float2 i##y##x##_2 = *(float2*)&p##y##x[ip];

						UNROLL10x3(LOAD_INPUT2);

#define LOAD_COEF(X)				float w_##X = w[X * 128];

#define CALC(IDX,Y,I0,I1,I2,I3,I4,I5,I6,I7)				\
						sum0 += w_##IDX * i##Y##I0; \
						sum1 += w_##IDX * i##Y##I1; \
						sum2 += w_##IDX * i##Y##I2; \
						sum3 += w_##IDX * i##Y##I3; \
						sum4 += w_##IDX * i##Y##I4; \
						sum5 += w_##IDX * i##Y##I5; \
						sum6 += w_##IDX * i##Y##I6; \
						sum7 += w_##IDX * i##Y##I7;


						{
#define LOAD_INPUT1X(Y,X)				float i##Y##X = i##Y##X##_2.x;

							UNROLL10x3(LOAD_INPUT1X);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

						ip++;
						{
#define LOAD_INPUT1Y(Y,X)				float i##Y##X = i##Y##X##_2.y;

							UNROLL10x3(LOAD_INPUT1Y);

							const float *w = (w0 + (ip * 128) * 9);
							UNROLL9(LOAD_COEF);

							{
								CALC(0,0,0,1,2,3,4,5,6,7);
								CALC(1,0,1,2,3,4,5,6,7,8);
								CALC(2,0,2,3,4,5,6,7,8,9);

								CALC(3,1,0,1,2,3,4,5,6,7);
								CALC(4,1,1,2,3,4,5,6,7,8);
								CALC(5,1,2,3,4,5,6,7,8,9);

								CALC(6,2,0,1,2,3,4,5,6,7);
								CALC(7,2,1,2,3,4,5,6,7,8);
								CALC(8,2,2,3,4,5,6,7,8,9);
							}
						}

					}

#define RELU(BI)							\
					{				\
						float *out = packed_output + (yi*wsz + (xi0+BI))*nOutputPlanes; \
									\
						{			\
							int opIndex = lid; \
							float v = sum##BI; \
							v += bv;	\
									\
							float mtz = max(v, 0.0f); \
							float ltz = min(v, 0.0f); \
									\
							v = ltz * 0.1f + mtz; \
									\
							out[opIndex] = v; \
						}			\
					}

					UNROLL8(RELU);
				}
			} else {
				for (int bi=0; bi<BLOCK_SIZE; bi++) {
					int xi = xi0+bi;
					if (xi == wsz) {
						break;
					}

					const float *w0 = weight + lid;
					float sum = 0;

					for (int ip=0; ip<nInputPlanes; ip++) {
						float i00, i01, i02;
						float i10, i11, i12;
						float i20, i21, i22;

						i00 = in_block0[(bi-1)*nInputPlanes+ip];
						i10 = in_block1[(bi-1)*nInputPlanes+ip];
						i20 = in_block2[(bi-1)*nInputPlanes+ip];

						i01 = in_block0[bi*nInputPlanes+ip];
						i11 = in_block1[bi*nInputPlanes+ip];
						i21 = in_block2[bi*nInputPlanes+ip];

						i02 = in_block0[(bi+1)*nInputPlanes+ip];
						i12 = in_block1[(bi+1)*nInputPlanes+ip];
						i22 = in_block2[(bi+1)*nInputPlanes+ip];

						const float *w = w0;
						sum += w[(9*ip+0) * 128]*i00;
						sum += w[(9*ip+1) * 128]*i01;
						sum += w[(9*ip+2) * 128]*i02;

						sum += w[(9*ip+3) * 128]*i10;
						sum += w[(9*ip+4) * 128]*i11;
						sum += w[(9*ip+5) * 128]*i12;

						sum += w[(9*ip+6) * 128]*i20;
						sum += w[(9*ip+7) * 128]*i21;
						sum += w[(9*ip+8) * 128]*i22;
					}

					float *out = packed_output + (yi*wsz + xi)*nOutputPlanes;
					{
						float v = sum;
						v += bv;

						float mtz = max(v, 0.0f);
						float ltz = min(v, 0.0f);

						v = ltz * 0.1f + mtz;
						out[op] = v;
					}
				}
			}
		}
	}
}

